#include "mmul.h"

void mmul(hipblasHandle_t handle, const double *A, const double *B, double *C, int n)
{
    const double alpha = (double)1.0;
    const double beta = (double)1.0;
    hipblasDgemm(handle,
                HIPBLAS_OP_N, HIPBLAS_OP_N,
                n, n, n,
                &alpha,
                A, n,
                B, n,
                &beta,
                C, n);
    hipDeviceSynchronize();
    return;
}